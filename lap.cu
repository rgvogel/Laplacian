/*
 * dotproduct.cu
 * includes setup funtion called from "driver" program
 * also includes kernel function 'kernel_dotproduct[2]()'
 * largely inspired in the pdf http://www.cuvilib.com/Reduction.pdf
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define BLOCK_SIZE 1024

struct timeval  tp1, tp2;
#define GPU_ERR_CHK(ans) { gpu_assert((ans), __FILE__, __LINE__); }
static void gpu_assert(hipError_t code, const char *file, int line,
        bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code),
                file, line);
        if (abort) {
            exit(code);
        }
    }
}
__global__ void kernel_laplacian(float *lap,long long size, int tUnit) {
    extern __shared__ float lapd[];
    
    int bid = blockDim.x;
    int nTotalThreads;
    nTotalThreads= 0;
    if (!bid){
	nTotalThreads = bid;
    }else{
	//(0 == 2^0)
    	int x = 1;
    	while(x < bid)
    	{
      	    x <<= 1;
    	}
        nTotalThreads = x;
    }

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    long long i = blockIdx.x*nTotalThreads + threadIdx.x;
    lapd[tid] = 1;
    if(i < size){
    	lapd[tid]= (lap[i-1]+lap[i+1])/2;
	if(i==(size-1)){
    		lapd[tid]= (lap[i-1]+23)/2;
	}
    }
    __syncthreads();
    if(i && i < size){
    	lap[i] = lapd[tid];}
   
}



// This function is called from the host computer.
// It manages memory and calls the function that is executed on the GPU
extern "C" void cuda_laplacian(float *lap,long long arraySize, int tUnit, double*time_result)
{
	// force_d, distance_d and result_d are the GPU counterparts of the arrays that exists in host memory 
	float *lap_d;
	

	

	// Reset the device and exit
    	GPU_ERR_CHK(hipDeviceReset());

    		
	// allocate space in the device 
	GPU_ERR_CHK(hipMalloc ((void**) &lap_d, sizeof(float) * arraySize));
        
	
	//copy the arrays from host to the device 
	GPU_ERR_CHK(hipMemcpy (lap_d, lap, sizeof(float) * arraySize, hipMemcpyHostToDevice));

	
	int threads;
	if(arraySize < 128){
		threads = 64;
	} else if (arraySize < 256 ){
		threads = 128;
	} else if (arraySize < 512){
		threads = 256;
	} else if (arraySize < 1024){
		threads = 512;
	} else {
		threads = BLOCK_SIZE;
	}
	long long block_size = threads;
        long long blocks = ceil(arraySize / ((float) block_size));
	// set execution configuration
        dim3 dimblock (block_size);
        dim3 dimgrid (blocks);
        int smemSize = dimblock.x * sizeof(long long);
        
	// actual computation: Call the kernel
	gettimeofday(&tp1, NULL);
	int i;
	for(i=0; i<tUnit;i++){
        kernel_laplacian<<<dimgrid,dimblock,smemSize>>>(lap_d, arraySize,tUnit);
                  
	} 
       //copy the arrays from host to the device
        GPU_ERR_CHK(hipMemcpy (lap, lap_d, sizeof(float) * arraySize, hipMemcpyDeviceToHost));
	gettimeofday(&tp2, NULL);
    	*time_result = (double) (tp2.tv_usec - tp1.tv_usec) / 1000000 + (double) (tp2.tv_sec - tp1.tv_sec);
       hipFree(lap_d); 
}

